#include <stdio.h>
#include <hip/hip_runtime.h>


extern "C"
{
extern void setCudaDevice(int myRank);
extern void matrixMultiplyChunk(int myRank, float* chunkData, float* b, float* chunkResult,
                            size_t numCellsInChunk, int threadsCount,
                            int m, int n, int k);
}

// Gets the cuda device count and sets the cuda device
// so that cuda is properly configured for the process
void setCudaDevice(int myRank) {
   int cE;
   int cudaDeviceCount;
   if ((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess) {
      printf(" Unable to determine cuda device count, error is %d, count is %d\n",
            cE, cudaDeviceCount);
      exit(-1);
   }
   if ((cE = hipSetDevice(myRank % cudaDeviceCount)) != hipSuccess) {
      printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
            myRank, (myRank % cudaDeviceCount), cE);
      exit(-1);
   }
}

// Uses CUDA threads to multiply matrices chunkData and b
// and stores results of multiplication chunkResult
// myRank - for debugging information
// m - number of rows of chunkData and chunkResult
// n - number of columns of b and chunkResult
// k - number of columns of chunkData and number of rows of b
__global__ void matrixMultiplyChunk_kernel(int myRank, float* chunkData, float* b, float* chunkResult,
                                        int m, int n, int k) {
    int device;
    hipGetDevice(&device);

    // hardcoded parallelized CUDA matrix multiply with CUDA threads
    size_t indexInChunk, z;
    size_t numCellsInChunk = m * k;
    for (indexInChunk = (blockIdx.x * blockDim.x) + threadIdx.x; indexInChunk < numCellsInChunk; indexInChunk += blockDim.x * gridDim.x) {
        size_t rowInChunk = indexInChunk / k;
        size_t rowOffsetChunk = rowInChunk * k;   // the offset to column 0 of current row in chunk
        size_t columnInChunk = indexInChunk - rowOffsetChunk;
        for (z = 0; z < k; z++) { // column offset of chunkData, row offset of b
            size_t rowOffsetb = z * k;   // the offset to column 0 of row z in b
            chunkResult[indexInChunk] += chunkData[rowOffsetChunk + z] * b[rowOffsetb + columnInChunk];
        }
    }
}

// Uses kernel function to execute matrix multiple of matrices chunkData and b
// and stores results in chunkResult
void matrixMultiplyChunk(int myRank, float* chunkData, float* b, float* chunkResult,
                        size_t numCellsInChunk, int threadsCount,
                        int m, int n, int k) {
    size_t numBlocks = (numCellsInChunk + threadsCount - 1) / threadsCount;
    matrixMultiplyChunk_kernel<<<numBlocks, threadsCount>>>(myRank, chunkData, b, chunkResult,
                                                            m, n, k);
    hipDeviceSynchronize();
}